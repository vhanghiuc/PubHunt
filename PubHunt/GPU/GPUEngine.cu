#include "hip/hip_runtime.h"
/*
 * This file is part of the VanitySearch distribution (https://github.com/JeanLucPons/VanitySearch).
 * Copyright (c) 2019 Jean Luc PONS.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#include "GPUEngine.h"
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <stdint.h>
#include "../Timer.h"

#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUCompute.h"

// ---------------------------------------------------------------------------------------

static const char* __cudaRandGetErrorEnum(hiprandStatus_t error) {
	switch (error) {
	case HIPRAND_STATUS_SUCCESS:
		return "HIPRAND_STATUS_SUCCESS";

	case HIPRAND_STATUS_VERSION_MISMATCH:
		return "HIPRAND_STATUS_VERSION_MISMATCH";

	case HIPRAND_STATUS_NOT_INITIALIZED:
		return "HIPRAND_STATUS_NOT_INITIALIZED";

	case HIPRAND_STATUS_ALLOCATION_FAILED:
		return "HIPRAND_STATUS_ALLOCATION_FAILED";

	case HIPRAND_STATUS_TYPE_ERROR:
		return "HIPRAND_STATUS_TYPE_ERROR";

	case HIPRAND_STATUS_OUT_OF_RANGE:
		return "HIPRAND_STATUS_OUT_OF_RANGE";

	case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
		return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";

	case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
		return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

	case HIPRAND_STATUS_LAUNCH_FAILURE:
		return "HIPRAND_STATUS_LAUNCH_FAILURE";

	case HIPRAND_STATUS_PREEXISTING_FAILURE:
		return "HIPRAND_STATUS_PREEXISTING_FAILURE";

	case HIPRAND_STATUS_INITIALIZATION_FAILED:
		return "HIPRAND_STATUS_INITIALIZATION_FAILED";

	case HIPRAND_STATUS_ARCH_MISMATCH:
		return "HIPRAND_STATUS_ARCH_MISMATCH";

	case HIPRAND_STATUS_INTERNAL_ERROR:
		return "HIPRAND_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}

inline void __cudaRandSafeCall(hiprandStatus_t err, const char* file, const int line)
{
	if (HIPRAND_STATUS_SUCCESS != err)
	{
		fprintf(stderr, "CudaRandSafeCall() failed at %s:%i : %s\n", file, line, __cudaRandGetErrorEnum(err));
		exit(-1);
	}
	return;
}

inline void __cudaSafeCall(hipError_t err, const char* file, const int line)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
		exit(-1);
	}
	return;
}

#define CudaRandSafeCall( err ) __cudaRandSafeCall( err, __FILE__, __LINE__ )
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

// ---------------------------------------------------------------------------------------

__global__ void compute_hash(uint64_t* keys, uint32_t* hash160, int numHash160, uint32_t maxFound, uint32_t* found)
{

	int id = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	ComputeHash(keys + id, hash160, numHash160, maxFound, found);

}

// ---------------------------------------------------------------------------------------

using namespace std;

int _ConvertSMVer2Cores(int major, int minor)
{

	// Defines for GPU Architecture types (using the SM version to determine
	// the # of cores per SM
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
		// and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
		{0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
		{0x30, 192},
		{0x32, 192},
		{0x35, 192},
		{0x37, 192},
		{0x50, 128},
		{0x52, 128},
		{0x53, 128},
		{0x60,  64},
		{0x61, 128},
		{0x62, 128},
		{0x70,  64},
		{0x72,  64},
		{0x75,  64},
		{0x80,  64},
		{0x86, 128},
		{0x89, 128},
		{-1, -1}
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	return 0;

}

// ----------------------------------------------------------------------------

GPUEngine::GPUEngine(int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound,
	const uint32_t* hash160, int numHash160)
{

	// Initialise CUDA
	this->nbThreadPerGroup = nbThreadPerGroup;
	this->numHash160 = numHash160;

	initialised = false;

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("GPUEngine: There are no available device(s) that support CUDA\n");
		exit(-1);
	}

	CudaSafeCall(hipSetDevice(gpuId));

	hipDeviceProp_t deviceProp;
	CudaSafeCall(hipGetDeviceProperties(&deviceProp, gpuId));

	if (nbThreadGroup == -1)
		nbThreadGroup = deviceProp.multiProcessorCount * 8;

	this->nbThread = nbThreadGroup * nbThreadPerGroup;
	this->maxFound = maxFound;
	this->outputSize = (maxFound * ITEM_SIZE_A + 4);

	char tmp[512];
	sprintf(tmp, "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
		gpuId, deviceProp.name, deviceProp.multiProcessorCount,
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		nbThread / nbThreadPerGroup,
		nbThreadPerGroup);
	deviceName = std::string(tmp);

	// Prefer L1 (We do not use __shared__ at all)
	CudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	size_t stackSize = 49152;
	CudaSafeCall(hipDeviceSetLimit(hipLimitStackSize, stackSize));

	// Allocate memory
	CudaSafeCall(hipMalloc((void**)&inputKey, nbThread * 4 * sizeof(uint64_t)));

	CudaSafeCall(hipMalloc((void**)&outputBuffer, outputSize));
	CudaSafeCall(hipHostAlloc(&outputBufferPinned, outputSize, hipHostMallocWriteCombined | hipHostMallocMapped));

	int K_SIZE = 5;

	CudaSafeCall(hipMalloc((void**)&inputHash, numHash160 * K_SIZE * sizeof(uint32_t)));
	CudaSafeCall(hipHostAlloc(&inputHashPinned, numHash160 * K_SIZE * sizeof(uint32_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	memcpy(inputHashPinned, hash160, numHash160 * K_SIZE * sizeof(uint32_t));

	CudaSafeCall(hipMemcpy(inputHash, inputHashPinned, numHash160 * K_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(inputHashPinned));
	inputHashPinned = NULL;

	// cuda-rand
	CudaSafeCall(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CudaRandSafeCall(hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64));
	CudaRandSafeCall(hiprandSetGeneratorOffset(prngGPU, std::time(0)));
	CudaRandSafeCall(hiprandSetStream(prngGPU, stream));

	Randomize();

	CudaSafeCall(hipGetLastError());

	initialised = true;

}

// ----------------------------------------------------------------------------

int GPUEngine::GetGroupSize()
{
	return GRP_SIZE;
}

// ----------------------------------------------------------------------------

void GPUEngine::PrintCudaInfo()
{
	const char* sComputeMode[] = {
		"Multiple host threads",
		"Only one host thread",
		"No host thread",
		"Multiple process threads",
		"Unknown",
		NULL
	};

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("GPUEngine: There are no available device(s) that support CUDA\n");
		return;
	}

	for (int i = 0; i < deviceCount; i++) {
		CudaSafeCall(hipSetDevice(i));
		hipDeviceProp_t deviceProp;
		CudaSafeCall(hipGetDeviceProperties(&deviceProp, i));
		printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
			i, deviceProp.name, deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			deviceProp.major, deviceProp.minor, (double)deviceProp.totalGlobalMem / 1048576.0,
			sComputeMode[deviceProp.computeMode]);
	}
}

// ----------------------------------------------------------------------------

GPUEngine::~GPUEngine()
{
	CudaSafeCall(hipFree(inputKey));
	CudaSafeCall(hipFree(inputHash));

	CudaSafeCall(hipHostFree(outputBufferPinned));
	CudaSafeCall(hipFree(outputBuffer));

	CudaRandSafeCall(hiprandDestroyGenerator(prngGPU));
	CudaSafeCall(hipStreamDestroy(stream));

}

// ----------------------------------------------------------------------------

int GPUEngine::GetNbThread()
{
	return nbThread;
}

// ----------------------------------------------------------------------------

bool GPUEngine::CallKernel()
{

	// Reset nbFound
	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	// Call the kernel (Perform STEP_SIZE keys per thread) 
	compute_hash << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
		(inputKey, inputHash, numHash160, maxFound, outputBuffer);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("GPUEngine: callKernel: %s\n", hipGetErrorString(err));
		return false;
	}
	return true;

}

// ----------------------------------------------------------------------------

bool GPUEngine::Step(std::vector<ITEM>& dataFound, bool spinWait)
{
	dataFound.clear();
	bool ret = true;

	ret = Randomize();

	ret = CallKernel();

	// Get the result
	if (spinWait) {
		CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, outputSize, hipMemcpyDeviceToHost));
	}
	else {
		// Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
		hipEvent_t evt;
		CudaSafeCall(hipEventCreate(&evt));
		CudaSafeCall(hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0));
		CudaSafeCall(hipEventRecord(evt, 0));
		while (hipEventQuery(evt) == hipErrorNotReady) {
			// Sleep 1 ms to free the CPU
			Timer::SleepMillis(1);
		}
		CudaSafeCall(hipEventDestroy(evt));
	}

	// Look for found
	uint32_t nbFound = outputBufferPinned[0];
	if (nbFound > maxFound) {
		nbFound = maxFound;
	}

	// When can perform a standard copy, the kernel is eneded
	CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE_A + 4, hipMemcpyDeviceToHost));

	for (uint32_t i = 0; i < nbFound; i++) {
		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE_A32 + 1);
		ITEM it;
		it.thId = itemPtr[0];
		it.pubKey = (uint8_t*)(itemPtr + 1);
		it.hash160 = (uint8_t*)(itemPtr + 10);
		dataFound.push_back(it);
	}

	return ret;
}

// ----------------------------------------------------------------------------

bool GPUEngine::Randomize()
{
	CudaRandSafeCall(hiprandGenerateLongLong(prngGPU, (unsigned long long*)inputKey, nbThread * 4));
	CudaSafeCall(hipStreamSynchronize(stream));

	return true;
}

// ----------------------------------------------------------------------------

